
#include <hip/hip_runtime.h>
#include <time.h>
#include <stdio.h>

#define THREADS 512

__device__ float* cQ = NULL;
__device__ float* cb_orig = NULL;

__device__ float* cb = NULL;

__device__ int* val = NULL;
__device__ int* best_val = NULL;

__device__ float sol;
__device__ float best_sol;

static __global__ void cuda_copy(float* t, float* s) {
	int i = threadIdx.x;
	t[i] = s[i];
}

static __global__ void cuda_zero_int(int* t) {
	int i = threadIdx.x;
	t[i] = 0;
}
static __global__ void cuda_changed(int idx, int n) {
	__shared__ int update;
	__shared__ float mult;
	int va;
	int i = threadIdx.x;
	if (i == 0) {
		va = val[idx];
		if (va == 0) {
			sol -= cb[idx];
			val[idx] = 1;
			mult = -1;
		} else {
			sol += cb[idx];
			val[idx] = 0;
			mult = 1;
		}
	}
	__syncthreads();
	cb[i] += mult*cQ[idx*n+i];
	if (i == 0) {
		update = 0;
		if (sol < best_sol) {
			best_sol = sol;
			update = 1;
		}
	}
	__syncthreads();
	if (update == 1) best_val[i] = val[i];
}
static void cuda_update(float* Q, float* b, int n) {
	hipMemcpy(cQ, Q, n*n*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(cb_orig, b, n*sizeof(float), hipMemcpyHostToDevice);
}
void cuda_initialize(int n) {
	hipMalloc((void**)&cQ, n*n*sizeof(float));
	hipMalloc((void**)&cb, n*sizeof(float));
	hipMalloc((void**)&cb_orig, n*sizeof(float));
	hipMalloc((void**)&val, n*sizeof(int));
	hipMalloc((void**)&best_val, n*sizeof(int));
}

void cuda_finalize() {
	hipFree(cQ);
	hipFree(cb);
	hipFree(cb_orig);
	hipFree(val);
	hipFree(best_val);
}

static __global__ void cuda_zero_sol() {
	sol = 0.;
	best_sol = 0.;
}

static void cuda_prepare_brute_force(int n) {
	cuda_copy<<< 1, n >>>(cb, cb_orig);
	cuda_zero_int<<< 1, n >>>(val);
	cuda_zero_int<<< 1, n >>>(best_val);
	cuda_zero_sol<<< 1, 1 >>>();
}
void cuda_brute_force(float* Q, float* b, int* ans, int n) {
	printf("Chamado com n = %d\n", n);
	clock_t before = clock();
	int lasti = 0;
	cuda_update(Q, b, n);
	cuda_prepare_brute_force(n);
	for (int _i = 1; _i < (1 << n); _i++) {
		int i, _changed, changed;
		i = _i ^ (_i >> 1);
		_changed = lasti ^ i;
		lasti = i;
		for (changed = -1; _changed; _changed >>= 1) changed++;
		if (_i % 1000 == 0) printf("Antes de chamar o cuda_changed %d!!\n", _i);
		cuda_changed<<<1, n>>>(changed, n);
	//	printf("Depois de chamar o cuda_changed!!\n");
	}
	hipMemcpy(ans, best_val, n*sizeof(int), hipMemcpyDeviceToHost);
	clock_t after = clock();
	printf("Cuda: Brute-force for %d vars in %lf secs\n", n, (1.*(after-before))/CLOCKS_PER_SEC);
}


